
#include <hip/hip_runtime.h>
///==========================================================
/// (v1d)
///		- Recompile with CUDA 7.5
///==========================================================

extern "C" void ConvGrid2SampSplitCD(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);

													
///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(double* dSDatR, double* dSDatI, double* dGDat, double* dKx, double* dKy, double* dKz, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
//double xF,yF,zF;
int xflr,yflr,zflr;
double DatValR,DatValI,KernVal,CrtValR,CrtValI;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = 0;
    DatValI = 0;	
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
    zflr = __double2int_rd(dKz[j]);   
	xF = lround(iKern*(dKx[j]-xflr));     	// lround necessary to match 'mex' and 'matlab'         
    yF = lround(iKern*(dKy[j]-yflr)); 
    zF = lround(iKern*(dKz[j]-zflr));	
	//xF = round(iKern*(dKx[j]-xflr));      
    //yF = round(iKern*(dKy[j]-yflr)); 
    //zF = round(iKern*(dKz[j]-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}
	for(c=-chW; c<=chW+1; c++) {
		for(b=-chW; b<=chW+1; b++) {
			for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernSz + lrintf(fabsf(zF-(c*iKern)))*KernSz*KernSz];
				CrtValR = dGDat[((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				CrtValI = dGDat[((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				DatValR += KernVal*CrtValR;
				DatValI += KernVal*CrtValI;				
				}
			}
		} 
	dSDatR[j] = DatValR;
	dSDatI[j] = DatValI;
	}

}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitCD(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDatR,*dSDatI,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDatR = (double*)(*HSampDatR+SampDatAdr*sizeof(double));
	dSDatI = (double*)(*HSampDatI+SampDatAdr*sizeof(double));
	dGDat = (double*)*HGrdDat;
	dKx = (double*)(*HKx+SampDatAdr*sizeof(double));
	dKy = (double*)(*HKy+SampDatAdr*sizeof(double));	
	dKz = (double*)(*HKz+SampDatAdr*sizeof(double));
	dKern = (double*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(DatLen)/double(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDatR,dSDatI,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished		
}
							
