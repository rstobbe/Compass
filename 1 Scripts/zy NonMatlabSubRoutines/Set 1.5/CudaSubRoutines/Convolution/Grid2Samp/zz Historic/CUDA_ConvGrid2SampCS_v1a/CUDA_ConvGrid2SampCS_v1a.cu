
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void ConvGrid2SampCS(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, char* Error);


///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDatR, float* dSDatI, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
float xrmd,yrmd,zrmd;
float fxflr,fyflr,fzflr;
int xflr,yflr,zflr;
int xF,yF,zF;
float DatValR,DatValI,KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = 0;
	DatValI = 0;
    fxflr = floorf(dKx[j]);
    fyflr = floorf(dKy[j]);
    fzflr = floorf(dKz[j]);
    xrmd = dKx[j]-fxflr;
    yrmd = dKy[j]-fyflr;
    zrmd = dKz[j]-fzflr;
    xF = lrintf(iKern*xrmd);                       
    yF = lrintf(iKern*yrmd); 
    zF = lrintf(iKern*zrmd);   
    if (xF == iKern){
        fxflr = fxflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        fyflr = fyflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        fzflr = fzflr + 1;
        zF = 0;
		}
	xflr = __float2int_rd(fxflr);
	yflr = __float2int_rd(fyflr);
	zflr = __float2int_rd(fzflr);	
    for(c=-chW; c<=chW+1; c++) {
        for(b=-chW; b<=chW+1; b++) {
            for(a=-chW; a<=chW+1; a++) {
                KernVal = dKern[lrintf(fabsf(xF-(a*iKern))+(fabsf(yF-(b*iKern))*KernSz)+(fabsf(zF-(c*iKern))*KernSz*KernSz))];
                DatValR += KernVal*dGDat[((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
                DatValI += KernVal*dGDat[((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				}
			}
		}  
	dSDatR[j] = DatValR;
	dSDatI[j] = DatValI;
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampCS(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, char* Error){

	float *dSDatR,*dSDatI,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDatR = (float*)*HSampDatR;
	dSDatI = (float*)*HSampDatI;
	dGDat = (float*)*HGrdDat;
	dKx = (float*)*HKx;
	dKy = (float*)*HKy;	
	dKz = (float*)*HKz;
	dKern = (float*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(DatLen)/float(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDatR,dSDatI,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

