
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void ConvGrid2SampRS(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, char* Error);


///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDat, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
float xrmd,yrmd,zrmd;
float fxflr,fyflr,fzflr;
int xflr,yflr,zflr;
int xF,yF,zF;
float DatVal,KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = 0;
    fxflr = floorf(dKx[j]);
    fyflr = floorf(dKy[j]);
    fzflr = floorf(dKz[j]);
    xrmd = dKx[j]-fxflr;
    yrmd = dKy[j]-fyflr;
    zrmd = dKz[j]-fzflr;
    xF = lrintf(iKern*xrmd);                       
    yF = lrintf(iKern*yrmd); 
    zF = lrintf(iKern*zrmd);   
    if (xF == iKern){
        fxflr = fxflr + 1;
        xF = 0;
	}
    if (yF == iKern){
        fyflr = fyflr + 1;
        yF = 0;
	}
    if (zF == iKern){
        fzflr = fzflr + 1;
        zF = 0;
	}
	xflr = __float2int_rd(fxflr);
	yflr = __float2int_rd(fyflr);
	zflr = __float2int_rd(fzflr);	
    for(c=-chW; c<=chW; c++) {
        for(b=-chW; b<=chW; b++) {
            for(a=-chW; a<=chW; a++) {
                KernVal = dKern[lrintf(fabsf(xF-(a*iKern))+(fabsf(yF-(b*iKern))*KernSz)+(fabsf(zF-(c*iKern))*KernSz*KernSz))];
                DatVal += KernVal*dGDat[(xflr+a-1)+((yflr+b-1)*GrdDatSz)+((zflr+c-1)*GrdDatSz*GrdDatSz)];
            }
        }
    }  
	dSDat[j] = DatVal;
}

}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampRS(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, char* Error){

	float *dSDat,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDat = (float*)*HSampDat;
	dGDat = (float*)*HGrdDat;
	dKx = (float*)*HKx;
	dKy = (float*)*HKy;	
	dKz = (float*)*HKz;
	dKern = (float*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(DatLen)/float(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							
