
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- RS_v1c start
///==========================================================

extern "C" void ConvGrid2SampSplitCD2D(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);

													
///=====================================================
/// Conv2D (kernel)					
///=====================================================
__global__ void Conv2D(double* dSDatR, double* dSDatI, double* dGDat, double* dKx, double* dKy, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF;
//double xF,yF;
int xflr,yflr;
double DatValR,DatValI,KernVal,CrtValR,CrtValI;
int j,a,b;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = 0;
    DatValI = 0;	
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]); 
	xF = lround(iKern*(dKx[j]-xflr));     	// lround necessary to match 'mex' and 'matlab'         
    yF = lround(iKern*(dKy[j]-yflr)); 
	//xF = round(iKern*(dKx[j]-xflr));      
    //yF = round(iKern*(dKy[j]-yflr)); 
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
	for(b=-chW; b<=chW+1; b++) {
		for(a=-chW; a<=chW+1; a++) {
			KernVal = dKern[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernSz];
			CrtValR = dGDat[((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)];
			CrtValI = dGDat[((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)];
			DatValR += KernVal*CrtValR;
			DatValI += KernVal*CrtValI;				
			}
		}
	dSDatR[j] = DatValR;
	dSDatI[j] = DatValI;
	}

}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitCD2D(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDatR,*dSDatI,*dGDat,*dKx,*dKy,*dKern;
	dSDatR = (double*)(*HSampDatR+SampDatAdr*sizeof(double));
	dSDatI = (double*)(*HSampDatI+SampDatAdr*sizeof(double));
	dGDat = (double*)*HGrdDat;
	dKx = (double*)(*HKx+SampDatAdr*sizeof(double));
	dKy = (double*)(*HKy+SampDatAdr*sizeof(double));	
	dKern = (double*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(DatLen)/double(tpb)));                           
	Conv2D<<<bpg,tpb>>>(dSDatR,dSDatI,dGDat,dKx,dKy,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished		
}
							
