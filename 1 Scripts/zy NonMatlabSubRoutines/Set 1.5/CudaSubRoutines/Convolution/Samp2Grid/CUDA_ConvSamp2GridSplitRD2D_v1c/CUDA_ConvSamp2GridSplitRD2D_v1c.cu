
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- CUDA_ConvSamp2GridSplitRD_v1c start
///==========================================================

extern "C" void ConvSamp2GridSplitRD2D(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);


__device__ double atomicAddD(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
		} while (assumed != old);
	return __longlong_as_double(old);
}							

///=====================================================
/// Conv2D (kernel)					
///=====================================================
__global__ void Conv2D(double* dSDat, double* dGDat, double* dKx, double* dKy, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF;
//double xF,yF;
int xflr,yflr;
double DatVal,KernVal;
int j,a,b;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = dSDat[j];
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
	xF = lround(iKern*(dKx[j]-xflr));     	// lround necessary to match 'mex' and 'matlab'         
    yF = lround(iKern*(dKy[j]-yflr)); 
	//xF = round(iKern*(dKx[j]-xflr));      
    //yF = round(iKern*(dKy[j]-yflr)); 
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}	
	for(b=-chW; b<=chW+1; b++) {
		for(a=-chW; a<=chW+1; a++) {
			KernVal = dKern[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernSz];		
			atomicAddD((dGDat+(xflr+a-1)+((yflr+b-1)*GrdDatSz)),(KernVal*DatVal));
			}
		}
	}  
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridSplitRD2D(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDat,*dGDat,*dKx,*dKy,*dKern;
	dSDat = (double*)(*HSampDat+SampDatAdr*sizeof(double));
	dGDat = (double*)*HGrdDat;
	dKx = (double*)(*HKx+SampDatAdr*sizeof(double));
	dKy = (double*)(*HKy+SampDatAdr*sizeof(double));	
	dKern = (double*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(DatLen)/double(tpb)));                           
	Conv2D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished	
}
							
