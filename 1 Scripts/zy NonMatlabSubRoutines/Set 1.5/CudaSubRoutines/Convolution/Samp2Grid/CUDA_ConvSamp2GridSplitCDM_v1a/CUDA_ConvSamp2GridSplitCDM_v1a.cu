
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- Use CUDA's atomicAdd (only available on compute 6.x and later)
///  	- switch to lrint
///		(still to-do: remove (j<DatLen) check)
///==========================================================

extern "C" void ConvSamp2GridSplitCDM(int Count, size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);
						

///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(double* dSDatR, double* dSDatI, double* dGDat, double* dKx, double* dKy, double* dKz, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
int xflr,yflr,zflr;
double DatValR,DatValI,KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = dSDatR[j];
    DatValI = dSDatI[j];	
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
    zflr = __double2int_rd(dKz[j]);   
	//xF = lround(iKern*(dKx[j]-xflr));     		// halfway rounded away from zero      
    //yF = lround(iKern*(dKy[j]-yflr)); 
    //zF = lround(iKern*(dKz[j]-zflr));	
	xF = lrint(iKern*(dKx[j]-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrint(iKern*(dKy[j]-yflr)); 
    zF = lrint(iKern*(dKz[j]-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}		
    for(c=-chW; c<=chW+1; c++) {
        for(b=-chW; b<=chW+1; b++) {
            for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernSz + lrint(fabsf(zF-(c*iKern)))*KernSz*KernSz];		
				atomicAdd((dGDat+((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)),(KernVal*DatValR));
				atomicAdd((dGDat+((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)),(KernVal*DatValI));
				}
			}
		}  
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridSplitCDM(int Count, size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDatR,*dSDatI,*dGDat,*dKx,*dKy,*dKz,*dKern;
	int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
    int PtsPerDevice = int(ceil(double(DatLen)/double(Count)));
	int bpg = int(ceil(double(PtsPerDevice)/double(tpb)));  
    for (int n=0; n<Count; n++){	
        hipSetDevice(n);
        dSDatR = (double*)(HSampDatR[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dSDatI = (double*)(HSampDatI[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dGDat = (double*)HGrdDat[n];
        dKx = (double*)(HKx[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dKy = (double*)(HKy[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));	
        dKz = (double*)(HKz[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dKern = (double*)HKern[n];	                
        Conv3D<<<bpg,tpb>>>(dSDatR,dSDatI,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,PtsPerDevice,KernSz,iKern,chW);
    }
	hipDeviceSynchronize();												// make sure finished	
	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}	
							
						
							